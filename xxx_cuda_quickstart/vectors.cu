/**
 * Making a minimal CUDA demo using snippets from documentation:
 *
 * https://docs.nvidia.com/cuda/cuda-quick-start-guide/index.html
 * https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#
 * https://www.cstechera.com/2015/07/addition-of-two-numbers-in-cuda-simple.html
 *
 * NOTES
 *
 * 1. CUDA kernels ~= shaders.
 * 2. But are written as an extension to CPP code, not in e.g. a shader language.
 *   * Declaration: A kernel is defined using the __global__ declaration specifier.
 *   * Invocation: n CUDA threads specified using a new <<<...>>>execution configuration syntax.
 * 3. Note that the official docs examples didn't have the necessary malloc and memcpy to<->from device
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

// The following sample code, using the built-in variable threadIdx, adds two vectors A and B of size N and stores the result into vector C.
__global__ void VecAdd( float* A, float* B, float* C ) {
  /*
For convenience, threadIdx is a 3-component vector, so that threads can be identified using a one-dimensional, two-dimensional, or three-dimensional thread
index, forming a one-dimensional, two-dimensional, or three-dimensional block of threads, called a thread block. This provides a natural way to invoke
computation across the elements in a domain such as a vector, matrix, or volume.

There is a limit to the number of threads per block, since all threads of a block are expected to reside on the same streaming multiprocessor core and must
share the limited memory resources of that core. On current GPUs, a thread block may contain up to 1024 threads.
  */
  int i = threadIdx.x;
  C[i]  = A[i] + B[i];
  printf( "C[%i] = %f\n", i, C[i] );
}

int main() {
  printf( "Hello CUDA\n" );

#define N 3
  float a[N] = { 1, 2, 3 };
  float b[N] = { 13, 22, 31 };
  float c[N] = { 0, 0, 0 };

  float *d_a, *d_b, *d_c; // Device variable Declaration

  // Allocation of Device Variables
  hipMalloc( (void**)&d_a, sizeof( float ) * 3 );
  hipMalloc( (void**)&d_b, sizeof( float ) * 3 );
  hipMalloc( (void**)&d_c, sizeof( float ) * 3 );

  // Copy Host Memory to Device Memory
  hipMemcpy( d_a, &a, sizeof( float ) * 3, hipMemcpyHostToDevice );
  hipMemcpy( d_b, &b, sizeof( float ) * 3, hipMemcpyHostToDevice );
  hipMemcpy( d_c, &c, sizeof( float ) * 3, hipMemcpyHostToDevice );

  // Kernel invocation with N threads
  // The number of threads per block and the number of blocks per grid specified in the <<<...>>> syntax can be of type int or dim3. Two-dimensional blocks or
  // grids can be specified as in the examples.
  VecAdd<<<1, N>>>( d_a, d_b, d_c );

  // Copy Device Memory to Host Memory
  hipMemcpy( &c, d_c, sizeof( float ) * 3, hipMemcpyDeviceToHost );

  printf( "before fence result C: %f %f %f\n", c[0], c[1], c[2] );

  // Fence all work
  hipDeviceSynchronize(); // can also call just to wait for 1 stream cudaStreamSynchronize(cudaStream)

  printf( "after frence result C: %f %f %f\n", c[0], c[1], c[2] );

  /* Another example:
// Kernel invocation with one block of N * N * 1 threads
int numBlocks = 1;
dim3 threadsPerBlock(N, N);
MatAdd<<<numBlocks, threadsPerBlock>>>(A, B, C);
  */

  printf( "Normal exit\n" );
  return 0;
}
