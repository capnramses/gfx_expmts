/**
 * Making a minimal CUDA demo using snippets from documentation:
 *
 * https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/
 *
 * NOTES
 *
 * 1. CUDA kernels ~= shaders.
 * 2. But are written as an extension to CPP code, not in e.g. a shader language.
 *   * Declaration: A kernel is defined using the __global__ declaration specifier.
 *   * Invocation: n CUDA threads specified using a new <<<...>>>execution configuration syntax.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    return 0;
}
